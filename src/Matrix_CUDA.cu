#include "hip/hip_runtime.h"
#include <math.h>

#include <iostream>

#include "Matrix_CUDA.cuh"

#define BLOCK_SIZE 16

namespace CUDA {

Matrix::Matrix(const ::Matrix& cpu_mat) : m_(cpu_mat.m_), n_(cpu_mat.n_) {
  allocateDeviceMemory();

  // Copy data from CPU to GPU
  hipError_t status = hipMemcpy(
      d_data, cpu_mat.data_, m_ * n_ * sizeof(float), hipMemcpyHostToDevice);

  if (status != hipSuccess) {
    freeDeviceMemory();
    throw std::runtime_error("Failed to copy data to GPU");
  }
}

Matrix::Matrix(const size_t m, const size_t n) : m_(m), n_(n) {
  allocateDeviceMemory();
}

Matrix::Matrix(const Matrix& matB) : m_(matB.m_), n_(matB.n_) {
  allocateDeviceMemory();

  // Copy data from GPU to GPU
  hipError_t status = hipMemcpy(d_data, matB.d_data, m_ * n_ * sizeof(float),
                                  hipMemcpyDeviceToDevice);

  if (status != hipSuccess) {
    freeDeviceMemory();
    throw std::runtime_error("Failed to copy data to GPU");
  }
}

Matrix::~Matrix() {
  m_ = 0;
  n_ = 0;
  freeDeviceMemory();
}

void Matrix::allocateDeviceMemory() {
  if (m_ * n_ > 0) {
    hipError_t status = hipMalloc(&d_data, m_ * n_ * sizeof(float));

    if (status != hipSuccess) {
      throw std::runtime_error("Failed to allocate GPU memory");
    }
  }
}

void Matrix::freeDeviceMemory() {
  if (d_data) {
    hipFree(d_data);
    d_data = nullptr;
  }
}

void Matrix::toCPU(::Matrix& cpu_mat) {
  cpu_mat = ::Matrix(m_, n_);

  hipError_t status = hipMemcpy(
      cpu_mat.data_, d_data, m_ * n_ * sizeof(float), hipMemcpyDeviceToHost);

  if (status != hipSuccess) {
    throw std::runtime_error("Failed to copy data back to CPU");
  }
}

Matrix& Matrix::operator=(const Matrix& matB) {
  freeDeviceMemory();

  m_ = matB.m_;
  n_ = matB.n_;

  allocateDeviceMemory();

  // Copy data from CPU to GPU
  hipError_t status = hipMemcpy(d_data, matB.d_data, m_ * n_ * sizeof(float),
                                  hipMemcpyDeviceToDevice);

  if (status != hipSuccess) {
    freeDeviceMemory();
    throw std::runtime_error("Failed to copy data to GPU");
  }

  return (*this);
}

// Example CUDA kernel for matrix multiplication
__global__ void matrixFillKernel(float* A, const float& val, size_t m,
                                 size_t n) {
  size_t row = blockIdx.x * blockDim.x + threadIdx.x;
  size_t col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < m && col < n) {
    A[row * n + col] = val;
  }
  __syncthreads();
}

void Matrix::fill(const float& val) {
  size_t m = m_, n = n_;

  // Set up grid and block dimensions
  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridDim((m + blockDim.x - 1) / blockDim.x,
               (n + blockDim.y - 1) / blockDim.y);

  // Launch kernel
  matrixFillKernel<<<gridDim, blockDim>>>(d_data, val, m, n);

  // Check for errors
  hipError_t status = hipGetLastError();
  if (status != hipSuccess) {
    throw std::runtime_error("Kernel launch failed");
  }

  // Synchronize
  hipDeviceSynchronize();
}

// Example CUDA kernel for matrix multiplication
__global__ void addKernel(const float* A, const float* B, float* output,
                          size_t m, size_t n) {
  size_t row = blockIdx.x * blockDim.x + threadIdx.x;
  size_t col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < m && col < n) {
    output[row * n + col] = A[row * n + col] + B[row * n + col];
  }
  __syncthreads();
}

Matrix& Matrix::add(const Matrix& matB) {
  size_t m = m_, n = n_;

  if (matB.m_ != m || matB.n_ != n) {
    throw std::logic_error("Size of A does not match size of B");
  }

  // Set up grid and block dimensions
  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridDim((m + blockDim.x - 1) / blockDim.x,
               (n + blockDim.y - 1) / blockDim.y);

  Matrix result(m, n);

  // Launch kernel
  addKernel<<<gridDim, blockDim>>>(d_data, matB.d_data, result.d_data, m, n);

  // Check for errors
  hipError_t status = hipGetLastError();
  if (status != hipSuccess) {
    throw std::runtime_error("Kernel launch failed");
  }

  // Synchronize
  hipDeviceSynchronize();

  return (*this) = result;
}

// Example CUDA kernel for matrix multiplication
__global__ void dotKernel(const float* A, const float* B, float* output,
                          size_t m, size_t k, size_t n) {
  size_t row = blockIdx.x * blockDim.x + threadIdx.x;
  size_t col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < m && col < n) {
    float sum = 0.0f;
    for (size_t i = 0; i < k; i++) {
      sum += A[k * row + i] * B[i * n + col];
    }
    output[row * n + col] = sum;
  }
  __syncthreads();
}

Matrix& Matrix::dot(const Matrix& matB) {
  size_t m = m_, k = n_, n = matB.n_;

  if (matB.m_ != k) {
    throw std::logic_error("Col # of A doesn't match Row # of B");
  }

  Matrix result(m, n);

  // Set up grid and block dimensions
  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridDim((m + blockDim.x - 1) / (blockDim.x * 1.0),
               (n + blockDim.y - 1) / (blockDim.y * 1.0));

  // Launch kernel
  dotKernel<<<gridDim, blockDim>>>(d_data, matB.d_data, result.d_data, m, k, n);

  // Check for errors
  hipError_t status = hipGetLastError();
  if (status != hipSuccess) {
    throw std::runtime_error("Kernel launch failed");
  }

  // Synchronize
  hipDeviceSynchronize();

  return (*this) = result;
}

// Example CUDA kernel for matrix multiplication
__global__ void convolutionKernel(const float* A, const float* mask,
                                  float* output, size_t m, size_t n, size_t k) {
  size_t row = blockIdx.x * blockDim.x + threadIdx.x;
  size_t col = blockIdx.y * blockDim.y + threadIdx.y;

  size_t out_m = m - k + 1;
  size_t out_n = n - k + 1;

  if (row < out_m && col < out_n) {
    float sum = 0.0f;
    for (size_t i = 0; i < k; i++) {
      for (size_t j = 0; j < k; j++) {
        // A[row + i][col + j] * mask[i][j]
        sum += A[(row + i) * n + (col + j)] * mask[i * k + j];
      }
    }
    output[row * out_n + col] = sum;
  }
  __syncthreads();
}

Matrix& Matrix::convolution(const Matrix& mask) {
  size_t m = m_, n = n_, k = mask.m_;

  if (mask.n_ != k) {
    throw std::logic_error("Mask is not a square matrix");
  }

  size_t out_m = m - k + 1;
  size_t out_n = n - k + 1;

  Matrix result(out_m, out_n);

  // Set up grid and block dimensions
  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridDim((out_m + blockDim.x - 1) / (blockDim.x * 1.0),
               (out_n + blockDim.y - 1) / (blockDim.y * 1.0));

  // Launch kernel
  convolutionKernel<<<gridDim, blockDim>>>(d_data, mask.d_data, result.d_data,
                                           m, n, k);

  // Check for errors
  hipError_t status = hipGetLastError();
  if (status != hipSuccess) {
    throw std::runtime_error("Kernel launch failed");
  }

  // Synchronize
  hipDeviceSynchronize();

  return (*this) = result;
}

}  // namespace CUDA